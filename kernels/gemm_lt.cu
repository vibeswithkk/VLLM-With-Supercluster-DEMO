#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipblaslt.h>
#include <iostream>
#include <memory>
#include <cstring>
#include <algorithm>
#include <cassert>

// Enterprise-grade error handling and logging
#define CUBLAS_SAFE_CALL(call) do { \
    hipblasStatus_t err = call; \
    if (err != HIPBLAS_STATUS_SUCCESS) { \
        fprintf(stderr, "cuBLAS error at %s:%d - %d\n", __FILE__, __LINE__, err); \
        return err; \
    } \
} while(0)

#define CUDA_SAFE_CALL(call) do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        return hipError_t(err); \
    } \
} while(0)

// Constants for performance optimization
constexpr size_t kDefaultWorkspaceSize = 256 * 1024 * 1024; // 256MB
constexpr size_t kMaxWorkspaceSize = 1024 * 1024 * 1024;    // 1GB
constexpr int kMaxAlgorithmHeuristics = 10;

// Enhanced GEMM configuration with improved resource management
struct GemmConfig {
    hipblasLtHandle_t ltHandle;
    hipblasLtMatmulDesc_t operationDesc;
    hipblasLtMatrixLayout_t Adesc, Bdesc, Cdesc, Ddesc;
    hipblasLtMatmulPreference_t preference;
    hipblasLtMatmulHeuristicResult_t heuristicResult;
    size_t workspaceSize;
    void* workspace;
    bool initialized;
    
    // Constructor
    GemmConfig() : ltHandle(nullptr), operationDesc(nullptr), 
                   Adesc(nullptr), Bdesc(nullptr), Cdesc(nullptr), Ddesc(nullptr),
                   preference(nullptr), workspaceSize(0), workspace(nullptr), initialized(false) {}
    
    // Destructor with proper cleanup
    ~GemmConfig() {
        cleanup();
    }
    
    // Cleanup resources
    void cleanup() {
        if (workspace) {
            hipFree(workspace);
            workspace = nullptr;
        }
        if (preference) {
            hipblasLtMatmulPreferenceDestroy(preference);
            preference = nullptr;
        }
        if (Ddesc) {
            hipblasLtMatrixLayoutDestroy(Ddesc);
            Ddesc = nullptr;
        }
        if (Cdesc) {
            hipblasLtMatrixLayoutDestroy(Cdesc);
            Cdesc = nullptr;
        }
        if (Bdesc) {
            hipblasLtMatrixLayoutDestroy(Bdesc);
            Bdesc = nullptr;
        }
        if (Adesc) {
            hipblasLtMatrixLayoutDestroy(Adesc);
            Adesc = nullptr;
        }
        if (operationDesc) {
            hipblasLtMatmulDescDestroy(operationDesc);
            operationDesc = nullptr;
        }
        if (ltHandle) {
            hipblasLtDestroy(ltHandle);
            ltHandle = nullptr;
        }
        initialized = false;
    }
};

// Initialize GEMM configuration with enhanced error handling
hipError_t init_gemm_config(GemmConfig& config, size_t workspace_size = kDefaultWorkspaceSize) {
    // Cleanup any existing resources
    config.cleanup();
    
    // Validate workspace size
    if (workspace_size == 0) {
        workspace_size = kDefaultWorkspaceSize;
    } else if (workspace_size > kMaxWorkspaceSize) {
        workspace_size = kMaxWorkspaceSize;
    }
    
    // Create cuBLASLt handle
    CUBLAS_SAFE_CALL(hipblasLtCreate(&config.ltHandle));
    
    // Create matrix multiplication descriptor
    CUBLAS_SAFE_CALL(hipblasLtMatmulDescCreate(&config.operationDesc, HIPBLAS_COMPUTE_32F_FAST_16F, HIP_R_32F));
    
    // Set transpose operations (no transpose for all matrices)
    hipblasOperation_t opTransA = HIPBLAS_OP_N;
    hipblasOperation_t opTransB = HIPBLAS_OP_N;
    CUBLAS_SAFE_CALL(hipblasLtMatmulDescSetAttribute(config.operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &opTransA, sizeof(opTransA)));
    CUBLAS_SAFE_CALL(hipblasLtMatmulDescSetAttribute(config.operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &opTransB, sizeof(opTransB)));
    
    // Set Epilogue to support bias addition
    hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_BIAS;
    CUBLAS_SAFE_CALL(hipblasLtMatmulDescSetAttribute(config.operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue)));
    
    // Create matrix layout descriptors
    CUBLAS_SAFE_CALL(hipblasLtMatrixLayoutCreate(&config.Adesc, HIP_R_16F, 0, 0, 0)); // Will be set in gemm_execute
    CUBLAS_SAFE_CALL(hipblasLtMatrixLayoutCreate(&config.Bdesc, HIP_R_16F, 0, 0, 0)); // Will be set in gemm_execute
    CUBLAS_SAFE_CALL(hipblasLtMatrixLayoutCreate(&config.Cdesc, HIP_R_32F, 0, 0, 0)); // Will be set in gemm_execute
    CUBLAS_SAFE_CALL(hipblasLtMatrixLayoutCreate(&config.Ddesc, HIP_R_32F, 0, 0, 0)); // Will be set in gemm_execute
    
    // Create preference handle
    CUBLAS_SAFE_CALL(hipblasLtMatmulPreferenceCreate(&config.preference));
    
    // Set workspace size
    config.workspaceSize = workspace_size;
    CUBLAS_SAFE_CALL(hipblasLtMatmulPreferenceSetAttribute(
        config.preference, 
        HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
        &config.workspaceSize, 
        sizeof(config.workspaceSize)
    ));
    
    // Allocate workspace
    CUDA_SAFE_CALL(hipMalloc(&config.workspace, config.workspaceSize));
    
    config.initialized = true;
    return hipSuccess;
}

// Execute GEMM operation: D = alpha * A * B + beta * C + bias
hipError_t gemm_execute(
    GemmConfig& config,
    float alpha,
    const void* A, hipDataType Atype, int64_t m, int64_t k, int64_t lda,
    const void* B, hipDataType Btype, int64_t k_, int64_t n, int64_t ldb,
    float beta,
    const void* C, hipDataType Ctype, int64_t m_, int64_t n_, int64_t ldc,
    void* D, hipDataType Dtype, int64_t m__, int64_t n__, int64_t ldd,
    const void* bias, hipDataType biasType, int64_t biasSize,
    hipStream_t stream
) {
    // Validate inputs
    if (!config.initialized) {
        return cudaErrorNotInitialized;
    }
    
    if (!A || !B || !C || !D) {
        return hipErrorInvalidValue;
    }
    
    if (m <= 0 || n <= 0 || k <= 0) {
        return hipErrorInvalidValue;
    }
    
    if (k != k_ || m != m_ || n != n_ || m != m__ || n != n__) {
        return hipErrorInvalidValue;
    }
    
    // Update matrix layout descriptors with current dimensions
    CUBLAS_SAFE_CALL(hipblasLtMatrixLayoutSetAttribute(config.Adesc, HIPBLASLT_MATRIX_LAYOUT_ROWS, &m, sizeof(m)));
    CUBLAS_SAFE_CALL(hipblasLtMatrixLayoutSetAttribute(config.Adesc, HIPBLASLT_MATRIX_LAYOUT_COLS, &k, sizeof(k)));
    CUBLAS_SAFE_CALL(hipblasLtMatrixLayoutSetAttribute(config.Adesc, HIPBLASLT_MATRIX_LAYOUT_LD, &lda, sizeof(lda)));
    CUBLAS_SAFE_CALL(hipblasLtMatrixLayoutSetAttribute(config.Adesc, HIPBLASLT_MATRIX_LAYOUT_TYPE, &Atype, sizeof(Atype)));
    
    CUBLAS_SAFE_CALL(hipblasLtMatrixLayoutSetAttribute(config.Bdesc, HIPBLASLT_MATRIX_LAYOUT_ROWS, &k, sizeof(k)));
    CUBLAS_SAFE_CALL(hipblasLtMatrixLayoutSetAttribute(config.Bdesc, HIPBLASLT_MATRIX_LAYOUT_COLS, &n, sizeof(n)));
    CUBLAS_SAFE_CALL(hipblasLtMatrixLayoutSetAttribute(config.Bdesc, HIPBLASLT_MATRIX_LAYOUT_LD, &ldb, sizeof(ldb)));
    CUBLAS_SAFE_CALL(hipblasLtMatrixLayoutSetAttribute(config.Bdesc, HIPBLASLT_MATRIX_LAYOUT_TYPE, &Btype, sizeof(Btype)));
    
    CUBLAS_SAFE_CALL(hipblasLtMatrixLayoutSetAttribute(config.Cdesc, HIPBLASLT_MATRIX_LAYOUT_ROWS, &m, sizeof(m)));
    CUBLAS_SAFE_CALL(hipblasLtMatrixLayoutSetAttribute(config.Cdesc, HIPBLASLT_MATRIX_LAYOUT_COLS, &n, sizeof(n)));
    CUBLAS_SAFE_CALL(hipblasLtMatrixLayoutSetAttribute(config.Cdesc, HIPBLASLT_MATRIX_LAYOUT_LD, &ldc, sizeof(ldc)));
    CUBLAS_SAFE_CALL(hipblasLtMatrixLayoutSetAttribute(config.Cdesc, HIPBLASLT_MATRIX_LAYOUT_TYPE, &Ctype, sizeof(Ctype)));
    
    CUBLAS_SAFE_CALL(hipblasLtMatrixLayoutSetAttribute(config.Ddesc, HIPBLASLT_MATRIX_LAYOUT_ROWS, &m, sizeof(m)));
    CUBLAS_SAFE_CALL(hipblasLtMatrixLayoutSetAttribute(config.Ddesc, HIPBLASLT_MATRIX_LAYOUT_COLS, &n, sizeof(n)));
    CUBLAS_SAFE_CALL(hipblasLtMatrixLayoutSetAttribute(config.Ddesc, HIPBLASLT_MATRIX_LAYOUT_LD, &ldd, sizeof(ldd)));
    CUBLAS_SAFE_CALL(hipblasLtMatrixLayoutSetAttribute(config.Ddesc, HIPBLASLT_MATRIX_LAYOUT_TYPE, &Dtype, sizeof(Dtype)));
    
    // Set bias if provided
    if (bias) {
        CUBLAS_SAFE_CALL(hipblasLtMatmulDescSetAttribute(config.operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof(bias)));
        CUBLAS_SAFE_CALL(hipblasLtMatmulDescSetAttribute(config.operationDesc, CUBLASLT_MATMUL_DESC_BIAS_TYPE, &biasType, sizeof(biasType)));
    }
    
    // Get algorithm heuristic
    int returnedResults = 0;
    CUBLAS_SAFE_CALL(hipblasLtMatmulAlgoGetHeuristic(
        config.ltHandle,
        config.operationDesc,
        config.Adesc,
        config.Bdesc,
        config.Cdesc,
        config.Ddesc,
        config.preference,
        kMaxAlgorithmHeuristics,
        &config.heuristicResult,
        &returnedResults
    ));
    
    if (returnedResults == 0) {
        fprintf(stderr, "No suitable algorithm found for GEMM operation\n");
        return hipErrorNotSupported;
    }
    
    // Set stream for the operation
    CUBLAS_SAFE_CALL(cublasLtSetStream(config.ltHandle, stream));
    
    // Execute matrix multiplication
    CUBLAS_SAFE_CALL(hipblasLtMatmul(
        config.ltHandle,
        config.operationDesc,
        &alpha,
        A,
        config.Adesc,
        B,
        config.Bdesc,
        &beta,
        C,
        config.Cdesc,
        D,
        config.Ddesc,
        &config.heuristicResult.algo,
        config.workspace,
        config.workspaceSize,
        stream
    ));
    
    return hipSuccess;
}

// Specialized GEMM for common FP16 x FP16 -> FP32 operations
hipError_t gemm_fp16_fp32(
    GemmConfig& config,
    float alpha,
    const __half* A, int64_t m, int64_t k, int64_t lda,
    const __half* B, int64_t k_, int64_t n, int64_t ldb,
    float beta,
    const float* C, int64_t m_, int64_t n_, int64_t ldc,
    float* D, int64_t m__, int64_t n__, int64_t ldd,
    const float* bias, int64_t biasSize,
    hipStream_t stream
) {
    return gemm_execute(
        config,
        alpha,
        A, HIP_R_16F, m, k, lda,
        B, HIP_R_16F, k_, n, ldb,
        beta,
        C, HIP_R_32F, m_, n_, ldc,
        D, HIP_R_32F, m__, n__, ldd,
        bias, HIP_R_32F, biasSize,
        stream
    );
}

// Batched GEMM operation for processing multiple matrices
hipError_t gemm_batched(
    GemmConfig& config,
    float alpha,
    const void* A[], hipDataType Atype, int64_t m, int64_t k, int64_t lda,
    const void* B[], hipDataType Btype, int64_t k_, int64_t n, int64_t ldb,
    float beta,
    const void* C[], hipDataType Ctype, int64_t m_, int64_t n_, int64_t ldc,
    void* D[], hipDataType Dtype, int64_t m__, int64_t n__, int64_t ldd,
    int64_t batchCount,
    hipStream_t stream
) {
    // Validate inputs
    if (!config.initialized) {
        return cudaErrorNotInitialized;
    }
    
    if (!A || !B || !C || !D) {
        return hipErrorInvalidValue;
    }
    
    if (batchCount <= 0) {
        return hipErrorInvalidValue;
    }
    
    // For batched operations, we'll use a loop for now
    // In a production system, we'd use cublasLtMatmulStridedBatched
    for (int64_t i = 0; i < batchCount; ++i) {
        hipError_t err = gemm_execute(
            config,
            alpha,
            A[i], Atype, m, k, lda,
            B[i], Btype, k_, n, ldb,
            beta,
            C[i], Ctype, m_, n_, ldc,
            D[i], Dtype, m__, n__, ldd,
            nullptr, HIP_R_32F, 0,
            stream
        );
        
        if (err != hipSuccess) {
            return err;
        }
    }
    
    return hipSuccess;
}

// Query available algorithms for a given configuration
int query_algorithms(
    GemmConfig& config,
    const void* A, hipDataType Atype, int64_t m, int64_t k, int64_t lda,
    const void* B, hipDataType Btype, int64_t k_, int64_t n, int64_t ldb,
    const void* C, hipDataType Ctype, int64_t m_, int64_t n_, int64_t ldc,
    void* D, hipDataType Dtype, int64_t m__, int64_t n__, int64_t ldd,
    hipblasLtMatmulHeuristicResult_t results[],
    int maxResults
) {
    if (!config.initialized || !results || maxResults <= 0) {
        return 0;
    }
    
    // Update matrix layout descriptors
    hipblasLtMatrixLayoutSetAttribute(config.Adesc, HIPBLASLT_MATRIX_LAYOUT_ROWS, &m, sizeof(m));
    hipblasLtMatrixLayoutSetAttribute(config.Adesc, HIPBLASLT_MATRIX_LAYOUT_COLS, &k, sizeof(k));
    hipblasLtMatrixLayoutSetAttribute(config.Adesc, HIPBLASLT_MATRIX_LAYOUT_LD, &lda, sizeof(lda));
    hipblasLtMatrixLayoutSetAttribute(config.Adesc, HIPBLASLT_MATRIX_LAYOUT_TYPE, &Atype, sizeof(Atype));
    
    hipblasLtMatrixLayoutSetAttribute(config.Bdesc, HIPBLASLT_MATRIX_LAYOUT_ROWS, &k, sizeof(k));
    hipblasLtMatrixLayoutSetAttribute(config.Bdesc, HIPBLASLT_MATRIX_LAYOUT_COLS, &n, sizeof(n));
    hipblasLtMatrixLayoutSetAttribute(config.Bdesc, HIPBLASLT_MATRIX_LAYOUT_LD, &ldb, sizeof(ldb));
    hipblasLtMatrixLayoutSetAttribute(config.Bdesc, HIPBLASLT_MATRIX_LAYOUT_TYPE, &Btype, sizeof(Btype));
    
    hipblasLtMatrixLayoutSetAttribute(config.Cdesc, HIPBLASLT_MATRIX_LAYOUT_ROWS, &m, sizeof(m));
    hipblasLtMatrixLayoutSetAttribute(config.Cdesc, HIPBLASLT_MATRIX_LAYOUT_COLS, &n, sizeof(n));
    hipblasLtMatrixLayoutSetAttribute(config.Cdesc, HIPBLASLT_MATRIX_LAYOUT_LD, &ldc, sizeof(ldc));
    hipblasLtMatrixLayoutSetAttribute(config.Cdesc, HIPBLASLT_MATRIX_LAYOUT_TYPE, &Ctype, sizeof(Ctype));
    
    hipblasLtMatrixLayoutSetAttribute(config.Ddesc, HIPBLASLT_MATRIX_LAYOUT_ROWS, &m, sizeof(m));
    hipblasLtMatrixLayoutSetAttribute(config.Ddesc, HIPBLASLT_MATRIX_LAYOUT_COLS, &n, sizeof(n));
    hipblasLtMatrixLayoutSetAttribute(config.Ddesc, HIPBLASLT_MATRIX_LAYOUT_LD, &ldd, sizeof(ldd));
    hipblasLtMatrixLayoutSetAttribute(config.Ddesc, HIPBLASLT_MATRIX_LAYOUT_TYPE, &Dtype, sizeof(Dtype));
    
    // Get algorithm heuristic
    int returnedResults = 0;
    hipblasLtMatmulAlgoGetHeuristic(
        config.ltHandle,
        config.operationDesc,
        config.Adesc,
        config.Bdesc,
        config.Cdesc,
        config.Ddesc,
        config.preference,
        maxResults,
        results,
        &returnedResults
    );
    
    return returnedResults;
}